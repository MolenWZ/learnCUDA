#include <hip/hip_runtime.h>
#include <stdio.h>


void printMatrix(int *C, const int nx, const int ny)
{
    int *ic = C;
    printf("\nMatrix: (%d.%d)\n", nx, ny);

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            printf("%3d", ic[ix]);

        }

        ic += nx;
        printf("\n");
    }

    printf("\n");
    return;
}

__global__ void printThreadIndex(int *A, const int nx, const int ny)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    printf("thread_id (%d,%d) block_id (%d,%d) coordinate (%d,%d) Matrix index %2d element %2d\n", 
            threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y,ix, iy, idx, A[idx]);
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // get device information
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    // set matrix dimension
    int nx = 8;
    int ny = 6;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);

    // malloc host memory
    int *h_A;
    h_A = (int *)malloc(nBytes);

    // iniitialize host matrix with integer
    for (int i = 0; i < nxy; i++)
    {
        h_A[i] = i;
    }
    printMatrix(h_A, nx, ny);

    // malloc device memory
    int *d_MatA;
    hipMalloc((void **)&d_MatA, nBytes);

    // transfer data from host to device
    hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);

    // set up execution configuration
    dim3 block(4, 2);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // invoke the kernel
    printThreadIndex<<<grid, block>>>(d_MatA, nx, ny);
    hipGetLastError();

    // free host and devide memory
    hipFree(d_MatA);
    free(h_A);

    // reset device
    hipDeviceReset();

    return (0);
}
